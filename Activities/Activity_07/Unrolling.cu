#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define GPUErrorAssertion(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void unrolling2(int* input, int* temp, int size) {
    int tid = threadIdx.x;
    int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;
    int index = BLOCK_OFFSET + tid;

    int* i_data = input + BLOCK_OFFSET;

    if ((index + blockDim.x) < size) {
        input[index] += input[index + blockDim.x];
    }

    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset / 2) {
        if (tid < offset) {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0) {
        temp[blockIdx.x] = i_data[0];
    }


}

__global__ void transpose(int* input, int* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index_in = y * width + x;
        int index_out = x * height + y;
        output[index_out] = input[index_in];
    }
}

int main() {
    int data_size = 1 << 10;
    int byte_size = data_size * sizeof(int);
    int block_size = 32;
    int parallel_reduction = 2;

    int* h_input, * h_ref;
    h_input = (int*)malloc(byte_size);

    for (int i = 0; i < data_size; i++) h_input[i] = (double)(rand() % 10);

    dim3 block(block_size);
    dim3 grid((data_size / byte_size) / parallel_reduction);

    int temp = sizeof(int) * grid.x;
    h_ref = (int*)malloc(temp);

    int* d_input, * d_temp;

    hipMalloc((void**)&d_input, byte_size);
    hipMalloc((void**)&d_temp, temp);

    hipMemset(d_temp, 0, temp);
    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

    if (parallel_reduction == 2)
        unrolling2 << < grid, block >> > (d_ipnut, d_temp, data_size);

    hipDeviceSynchronize();
    hipMemcpy(h_ref, d_temp, temp, hipMemcpyDeviceToHost);

    int gpu_result = 0;
    for (int i = 0; i < grid.x; i++) {
        gpu_result += h_ref[i];
    }

    hipFree(d_input);
    hipFree(d_temp);
    free(h_input);
    free(h_ref);
    
    return 0;
}
