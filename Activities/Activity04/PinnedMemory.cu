#include "hip/hip_runtime.h"

#include "stdio.h"
#include "stdlib.h"
#include <iostream>

int main() {
	int size = 1 << 25;
	int bytes = size * sizeof(float);

	// Allocate the host memory
	float* h_a = (float *)malloc(bytes);

	// float* h_a;
	// cudaMallocHost((float**)&h_a, bytes);

	// Allocate the device memory
	float* d_a;
	hipMalloc((float**)&d_a, bytes);

	// initialize the host memory
	for (int i = 0; i < size; i++) {
		h_a[i] = rand() % 10;
	}
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

	// Free memory
	hipFree(d_a);
	free(h_a);
	// cudaFreeHost(h_a);


}
